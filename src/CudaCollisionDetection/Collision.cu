#include "hip/hip_runtime.h"
#include"Collision.cuh"
#include "Ball.hpp"

__device__ float Dist(float x, float y, float z)
{
	return sqrt(x * x + y * y + z * z);
}

__device__ float Dist(Point& p)
{
	return sqrt(p.x * p.x + p.y * p.y + p.z * p.z);
}

__device__ float Multiply(Point& a, Point& b)
{
	return (a.x * b.x + a.y * b.y + a.z * b.z);
}

/*
	������������߽���ײ
	������X��Χ��-X, X), Z��Χ(-Z, Z), Y��Χ(0, Y)
	���أ���
*/
__device__ void HandleCollisionBoard(Ball& ball, float XRange, float ZRange, float Height)
{
	if (ball.CurrentPlace.x - ball.Radius < -XRange)
	{
		ball.CurrentPlace.x = -XRange + ball.Radius;
		ball.CurrentSpeed.x = -ball.CurrentSpeed.x;
	}
	else if (ball.CurrentPlace.x + ball.Radius > XRange)
	{
		ball.CurrentPlace.x = XRange - ball.Radius;
		ball.CurrentSpeed.x = -ball.CurrentSpeed.x;
	}
	if (ball.CurrentPlace.z - ball.Radius < -ZRange)
	{
		ball.CurrentPlace.z = -ZRange + ball.Radius;
		ball.CurrentSpeed.z = -ball.CurrentSpeed.z;
	}
	else if (ball.CurrentPlace.z + ball.Radius > ZRange)
	{
		ball.CurrentPlace.z = ZRange - ball.Radius;
		ball.CurrentSpeed.z = -ball.CurrentSpeed.z;
	}
	if (ball.CurrentPlace.y - ball.Radius < 0)
	{
		ball.CurrentPlace.y = ball.Radius;
		ball.CurrentSpeed.y = -ball.CurrentSpeed.y;
	}
	else if (ball.CurrentPlace.y + ball.Radius > Height)
	{
		ball.CurrentPlace.y = Height - ball.Radius;
		ball.CurrentSpeed.y = -ball.CurrentSpeed.y;
	}
}



/*
	����������С�������˶�����߽���ײ
	�����������˶�ʱ�䣬X��Χ��-X, X), Z��Χ(-Z, Z), Y��Χ(0, Y)
	���أ���
*/
__device__ void BallMove(Ball& ball, float time, float XRange, float ZRange, float Height)
{

	ball.CurrentPlace.x = ball.CurrentPlace.x + ball.CurrentSpeed.x * time;
	ball.CurrentPlace.y = ball.CurrentPlace.y + ball.CurrentSpeed.y * time;
	ball.CurrentPlace.z = ball.CurrentPlace.z + ball.CurrentSpeed.z * time;
	HandleCollisionBoard(ball, XRange, ZRange, Height);
}

__global__ void UpdateBallsMove(Ball* balls, float TimeOnce, float XRange, float ZRange, float Height, int N)
{
	// ��ȡȫ������
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// ����
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < N; i += stride)
	{
		BallMove(balls[i], TimeOnce, XRange, ZRange, Height);
	}

}

/*
	�������ж��������Ƿ���ײ
	��������a����b
	���أ���1����0
*/
__device__ bool JudgeCollision(Ball& a, Ball& b)
{
	float dist = 0;
	float dist_x = a.CurrentPlace.x - b.CurrentPlace.x;
	float dist_y = a.CurrentPlace.y - b.CurrentPlace.y;
	float dist_z = a.CurrentPlace.z - b.CurrentPlace.z;
	dist = Dist(dist_x, dist_y, dist_z);
	if (dist < a.Radius + b.Radius)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

/*
	������������ײ������ٶ�
	��������a����b
	���أ���
*/
__device__ void ChangeSpeed(Ball& a, Ball& b)
{
	//�����ٶȰ����������任�������ٶȲ���
	float dist = 0;
	float diff_x = b.CurrentPlace.x - a.CurrentPlace.x;
	float diff_y = b.CurrentPlace.y - a.CurrentPlace.y;
	float diff_z = b.CurrentPlace.z - a.CurrentPlace.z;
	dist = Dist(diff_x, diff_y, diff_z);

	//���򣬷����ٶ�
	float rate_collide_a = (a.CurrentSpeed.x * diff_x + a.CurrentSpeed.y * diff_y + a.CurrentSpeed.z * diff_z) / dist / dist;
	float speed_collide_a_x = diff_x * rate_collide_a;
	float speed_collide_a_y = diff_y * rate_collide_a;
	float speed_collide_a_z = diff_z * rate_collide_a;

	float rate_collide_b = (b.CurrentSpeed.x * diff_x + b.CurrentSpeed.y * diff_y + b.CurrentSpeed.z * diff_z) / dist / dist;
	float speed_collide_b_x = diff_x * rate_collide_b;
	float speed_collide_b_y = diff_y * rate_collide_b;
	float speed_collide_b_z = diff_z * rate_collide_b;

	float unchanged_a_x = a.CurrentSpeed.x - speed_collide_a_x;
	float unchanged_a_y = a.CurrentSpeed.y - speed_collide_a_y;
	float unchanged_a_z = a.CurrentSpeed.z - speed_collide_a_z;

	float unchanged_b_x = b.CurrentSpeed.x - speed_collide_b_x;
	float unchanged_b_y = b.CurrentSpeed.y - speed_collide_b_y;
	float unchanged_b_z = b.CurrentSpeed.z - speed_collide_b_z;


	//����b������aײb���������߾����ٶ�
	float speed_collide_new_a_x = (speed_collide_a_x * (a.Weight - b.Weight) + speed_collide_b_x * (2 * b.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_a_y = (speed_collide_a_y * (a.Weight - b.Weight) + speed_collide_b_y * (2 * b.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_a_z = (speed_collide_a_z * (a.Weight - b.Weight) + speed_collide_b_z * (2 * b.Weight)) / (a.Weight + b.Weight);

	float speed_collide_new_b_x = (speed_collide_a_x * (2 * a.Weight) + speed_collide_b_x * (b.Weight - a.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_b_y = (speed_collide_a_y * (2 * a.Weight) + speed_collide_b_y * (b.Weight - a.Weight)) / (a.Weight + b.Weight);
	float speed_collide_new_b_z = (speed_collide_a_z * (2 * a.Weight) + speed_collide_b_z * (b.Weight - a.Weight)) / (a.Weight + b.Weight);

	a.CurrentSpeed.x = speed_collide_new_a_x + unchanged_a_x;
	a.CurrentSpeed.y = speed_collide_new_a_y + unchanged_a_y;
	a.CurrentSpeed.z = speed_collide_new_a_z + unchanged_a_z;

	b.CurrentSpeed.x = speed_collide_new_b_x + unchanged_b_x;
	b.CurrentSpeed.y = speed_collide_new_b_y + unchanged_b_y;
	b.CurrentSpeed.z = speed_collide_new_b_z + unchanged_b_z;
}

__global__ void HandleCollisionNaive(Ball* balls, int N)
{
	// ��ȡȫ������
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// ����
	int stride = blockDim.x * gridDim.x;
	for (int k = index; k < N * N; k += stride)
	{
		int i = k / N;
		int j = k % N;
		if(i < j)
		{
			if (JudgeCollision(balls[i], balls[j]))
			{
				ChangeSpeed(balls[i], balls[j]);
			}
		}
	}
}

void UpdateBallsNaiveGPU(Ball* balls, float TimeOnce, float XRange, float ZRange, float Height, int N)
{
	// �����й��ڴ�
	int nBytes = N * sizeof(Ball);
	Ball* balls_gpu;
	hipMallocManaged((void**)&balls_gpu, nBytes);

	// ��ʼ������
	hipMemcpy((void*)balls_gpu, (void*)balls, nBytes, hipMemcpyHostToDevice);

	// ����kernel��ִ������
	dim3 blockSize(256);
	dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

	// ִ��kernel
	HandleCollisionNaive << < gridSize, blockSize >> > (balls_gpu, N);
	// ͬ��device ��֤�������ȷ����
	hipDeviceSynchronize();

	// ִ��kernel
	UpdateBallsMove <<< gridSize, blockSize >>> (balls_gpu, TimeOnce, XRange, ZRange, Height, N);
	// ͬ��device ��֤�������ȷ����
	hipDeviceSynchronize();

	// ��¼���
	hipMemcpy((void*)balls, (void*)balls_gpu, nBytes, hipMemcpyDeviceToHost);

	// �ͷ��ڴ�
	hipFree(balls_gpu);
}